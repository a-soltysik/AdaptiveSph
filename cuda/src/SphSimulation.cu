#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <thrust/execution_policy.h>
#include <hip/hip_vector_types.h>

#include <cstddef>
#include <cstdint>
#include <cuda/Simulation.cuh>
#include <glm/common.hpp>
#include <glm/ext/vector_float3.hpp>
#include <glm/ext/vector_uint3.hpp>
#include <memory>
#include <thrust/detail/device_ptr.inl>
#include <thrust/detail/sort.inl>
#include <type_traits>
#include <vector>

#include "Algorithm.cuh"
#include "ImportedParticleMemory.cuh"
#include "Span.cuh"
#include "SphSimulation.cuh"
#include "glm/ext/scalar_constants.hpp"

namespace sph::cuda
{

SphSimulation::SphSimulation(const Parameters& initialParameters,
                             const std::vector<glm::vec4>& positions,
                             const ParticlesDataBuffer& memory)
    : _particleBuffer {toInternalBuffer(memory)},
      _simulationData {initialParameters},
      _state {.grid = createGrid(initialParameters, positions.size())},
      _particleCount {static_cast<uint32_t>(positions.size())}
{
    const auto velocitiesVec = std::vector(positions.size(), glm::vec4(0.0f));
    const auto radiusesVec = std::vector(positions.size(), initialParameters.particleRadius);
    auto massesVec = std::vector(
        positions.size(),
        getParticleMass(initialParameters.domain.getVolume(), initialParameters.restDensity, _particleCount));

    hipMemcpy(_particleBuffer.positions.getData<glm::vec4>(),
               positions.data(),
               positions.size() * sizeof(glm::vec4),
               hipMemcpyHostToDevice);

    hipMemcpy(_particleBuffer.velocities.getData<glm::vec4>(),
               velocitiesVec.data(),
               velocitiesVec.size() * sizeof(glm::vec4),
               hipMemcpyHostToDevice);

    hipMemcpy(_particleBuffer.predictedPositions.getData<glm::vec4>(),
               positions.data(),
               positions.size() * sizeof(glm::vec4),
               hipMemcpyHostToDevice);

    hipMemcpy(_particleBuffer.radiuses.getData<float>(),
               radiusesVec.data(),
               radiusesVec.size() * sizeof(float),
               hipMemcpyHostToDevice);

    hipMemcpy(_particleBuffer.masses.getData<float>(),
               massesVec.data(),
               massesVec.size() * sizeof(float),
               hipMemcpyHostToDevice);
}

SphSimulation::~SphSimulation()
{
    hipFree(_state.grid.particleGridIndices.data);
    hipFree(_state.grid.particleArrayIndices.data);
    hipFree(_state.grid.cellStartIndices.data);
    hipFree(_state.grid.cellEndIndices.data);
}

auto SphSimulation::toInternalBuffer(const ParticlesDataBuffer& memory) -> ParticlesInternalDataBuffer
{
    return {.positions = dynamic_cast<const ImportedParticleMemory&>(memory.positions),
            .predictedPositions = dynamic_cast<const ImportedParticleMemory&>(memory.predictedPositions),
            .velocities = dynamic_cast<const ImportedParticleMemory&>(memory.velocities),
            .forces = dynamic_cast<const ImportedParticleMemory&>(memory.forces),
            .densities = dynamic_cast<const ImportedParticleMemory&>(memory.densities),
            .nearDensities = dynamic_cast<const ImportedParticleMemory&>(memory.nearDensities),
            .pressures = dynamic_cast<const ImportedParticleMemory&>(memory.pressures),
            .radiuses = dynamic_cast<const ImportedParticleMemory&>(memory.radiuses),
            .masses = dynamic_cast<const ImportedParticleMemory&>(memory.masses)};
}

void SphSimulation::update(const Parameters& parameters, float deltaTime)
{
    _simulationData = parameters;

    computeExternalForces(deltaTime);
    resetGrid();
    assignParticlesToCells();
    sortParticles();
    calculateCellStartAndEndIndices();
    computeDensities();
    computePressureForce(deltaTime);
    computeViscosityForce(deltaTime);
    integrateMotion(deltaTime);
    handleCollisions();

    hipDeviceSynchronize();
}

auto createSimulation(const Simulation::Parameters& data,
                      const std::vector<glm::vec4>& positions,
                      const ParticlesDataBuffer& memory) -> std::unique_ptr<Simulation>
{
    return std::make_unique<SphSimulation>(data, positions, memory);
}

auto SphSimulation::createGrid(const Parameters& data, size_t particleCount) -> Grid
{
    int32_t* particleIndices {};
    int32_t* particleArrayIndices {};
    int32_t* cellStartIndices {};
    int32_t* cellEndIndices {};

    const auto gridCellWidth = 2 * data.smoothingRadius;
    const auto gridCellCount = glm::uvec3 {glm::ceil((data.domain.max - data.domain.min) / gridCellWidth)};

    hipMalloc(reinterpret_cast<void**>(&particleIndices), particleCount * sizeof(int32_t));
    hipMalloc(reinterpret_cast<void**>(&particleArrayIndices), particleCount * sizeof(int32_t));
    hipMalloc(reinterpret_cast<void**>(&cellStartIndices),
               gridCellCount.x * gridCellCount.y * gridCellCount.z * sizeof(int32_t));
    hipMalloc(reinterpret_cast<void**>(&cellEndIndices),
               gridCellCount.x * gridCellCount.y * gridCellCount.z * sizeof(int32_t));

    return Grid {
        .gridSize = gridCellCount,
        .cellSize = glm::vec3 {gridCellWidth},
        .cellStartIndices =
            Span {.data = cellStartIndices, .size = gridCellCount.x * gridCellCount.y * gridCellCount.z},
        .cellEndIndices = Span {.data = cellEndIndices, .size = gridCellCount.x * gridCellCount.y * gridCellCount.z},
        .particleGridIndices = Span {.data = particleIndices, .size = particleCount},
        .particleArrayIndices = Span {.data = particleArrayIndices, .size = particleCount}
    };
}

auto SphSimulation::getBlocksPerGridForParticles() const -> dim3
{
    return {(_particleCount + _simulationData.threadsPerBlock - 1) / _simulationData.threadsPerBlock};
}

auto SphSimulation::getBlocksPerGridForGrid() const -> dim3
{
    return {(_state.grid.gridSize.x * _state.grid.gridSize.y * _state.grid.gridSize.z +
             _simulationData.threadsPerBlock - 1) /
            _simulationData.threadsPerBlock};
}

auto SphSimulation::getParticles() const -> ParticlesData
{
    return {
        .positions = _particleBuffer.positions.getData<std::remove_pointer_t<decltype(ParticlesData::positions)>>(),
        .predictedPositions = _particleBuffer.predictedPositions
                                  .getData<std::remove_pointer_t<decltype(ParticlesData::predictedPositions)>>(),
        .velocities = _particleBuffer.velocities.getData<std::remove_pointer_t<decltype(ParticlesData::velocities)>>(),
        .forces = _particleBuffer.forces.getData<std::remove_pointer_t<decltype(ParticlesData::forces)>>(),
        .densities = _particleBuffer.densities.getData<std::remove_pointer_t<decltype(ParticlesData::densities)>>(),
        .nearDensities =
            _particleBuffer.nearDensities.getData<std::remove_pointer_t<decltype(ParticlesData::nearDensities)>>(),
        .pressures = _particleBuffer.pressures.getData<std::remove_pointer_t<decltype(ParticlesData::pressures)>>(),
        .radiuses = _particleBuffer.radiuses.getData<std::remove_pointer_t<decltype(ParticlesData::radiuses)>>(),
        .masses = _particleBuffer.masses.getData<std::remove_pointer_t<decltype(ParticlesData::masses)>>(),
        .particleCount = _particleCount};
}

void SphSimulation::computeExternalForces(float deltaTime) const
{
    kernel::computeExternalForces<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(getParticles(),
                                                                                                       _simulationData,
                                                                                                       deltaTime);
}

void SphSimulation::resetGrid() const
{
    kernel::resetGrid<<<getBlocksPerGridForGrid(), _simulationData.threadsPerBlock>>>(_state.grid);
}

void SphSimulation::assignParticlesToCells() const
{
    kernel::assignParticlesToCells<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(
        getParticles(),
        _state,
        _simulationData);
}

void SphSimulation::sortParticles() const
{
    thrust::sort_by_key(
        thrust::device,
        thrust::device_pointer_cast(_state.grid.particleGridIndices.data),
        thrust::device_pointer_cast(_state.grid.particleGridIndices.data + _state.grid.particleGridIndices.size),
        thrust::device_pointer_cast(_state.grid.particleArrayIndices.data));
}

void SphSimulation::calculateCellStartAndEndIndices() const
{
    kernel::calculateCellStartAndEndIndices<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(
        _state.grid);
}

void SphSimulation::computeDensities() const
{
    kernel::computeDensities<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(getParticles(),
                                                                                                  _state,
                                                                                                  _simulationData);
}

void SphSimulation::computePressureForce(float deltaTime) const
{
    kernel::computePressureForce<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(getParticles(),
                                                                                                      _state,
                                                                                                      _simulationData,
                                                                                                      deltaTime);
}

void SphSimulation::computeViscosityForce(float deltaTime) const
{
    kernel::computeViscosityForce<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(getParticles(),
                                                                                                       _state,
                                                                                                       _simulationData,
                                                                                                       deltaTime);
}

void SphSimulation::integrateMotion(float deltaTime) const
{
    kernel::integrateMotion<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(getParticles(),
                                                                                                 _simulationData,
                                                                                                 deltaTime);
}

void SphSimulation::handleCollisions() const
{
    kernel::handleCollisions<<<getBlocksPerGridForParticles(), _simulationData.threadsPerBlock>>>(getParticles(),
                                                                                                  _simulationData);
}

auto SphSimulation::getParticleMass(float domainVolume, float restDensity, uint32_t particlesCount) -> float
{
    return domainVolume * restDensity / particlesCount;
}

}
