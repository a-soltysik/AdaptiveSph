#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <hip/hip_vector_types.h>

#include <cstddef>
#include <cstdint>
#include <vector>

#include "AdaptiveSphSimulation.cuh"
#include "Span.cuh"
#include "SphSimulation.cuh"
#include "common/Utils.cuh"
#include "cuda/Simulation.cuh"
#include "cuda/refinement/RefinementParameters.cuh"
#include "refinement/Common.cuh"
#include "refinement/CurvatureCriterion.cuh"
#include "refinement/InterfaceCriterion.cuh"
#include "refinement/ParticleOperations.cuh"
#include "refinement/VelocityCriterion.cuh"
#include "refinement/VorticityCriterion.cuh"

namespace sph::cuda
{

template <typename CriterionGenerator>
__global__ void getCriterionValuesWithGrid(ParticlesData particles,
                                           Span<float> splitCriterionValues,
                                           CriterionGenerator criterionGenerator,
                                           const SphSimulation::Grid grid,
                                           const Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }
    const auto value = criterionGenerator(particles, idx, grid, simulationData);
    splitCriterionValues.data[idx] = value;
}

template <typename CriterionGenerator>
__global__ void getCriterionValuesNoGrid(ParticlesData particles,
                                         Span<float> splitCriterionValues,
                                         CriterionGenerator criterionGenerator,
                                         const Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }
    const auto value = criterionGenerator(particles, idx, simulationData);
    splitCriterionValues.data[idx] = value;
}

AdaptiveSphSimulation::AdaptiveSphSimulation(const Parameters& initialParameters,
                                             const std::vector<glm::vec4>& positions,
                                             const ParticlesDataBuffer& memory,
                                             const refinement::RefinementParameters& refinementParams)
    : SphSimulation(initialParameters, positions, memory, refinementParams.maxParticleCount),
      _refinementParams(refinementParams),
      _refinementData {initializeRefinementData(_refinementParams.maxParticleCount, _refinementParams.maxBatchRatio)},
      _enhancedMergeData {allocateEnhancedMergeData(refinementParams.maxParticleCount)},
      _targetParticleCount {static_cast<uint32_t>(positions.size())}
{
    const auto initialCount = SphSimulation::getParticlesCount();
    hipMemcpy(_refinementData.particlesCount, &initialCount, sizeof(uint32_t), hipMemcpyHostToDevice);
}

AdaptiveSphSimulation::~AdaptiveSphSimulation()
{
    hipFree(_refinementData.split.criterionValues.data);
    hipFree(_refinementData.split.particlesIdsToSplit.data);
    hipFree(_refinementData.split.particlesSplitCount);

    // Free merge-related memory
    hipFree(_refinementData.merge.criterionValues.data);
    hipFree(_refinementData.merge.particlesIdsToMerge.first.data);
    hipFree(_refinementData.merge.particlesIdsToMerge.second.data);
    hipFree(_refinementData.merge.removalFlags.data);
    hipFree(_refinementData.merge.prefixSums.data);
    hipFree(_refinementData.merge.particlesMergeCount);

    // Free shared memory
    hipFree(_refinementData.particlesIds.data);
    hipFree(_refinementData.particlesCount);

    freeEnhancedMergeData(_enhancedMergeData);
}

auto AdaptiveSphSimulation::initializeRefinementData(uint32_t maxParticleCount, float maxBatchSize)
    -> refinement::RefinementData
{
    uint32_t* particlesIdsToSplit = nullptr;
    uint32_t* particlesIds = nullptr;
    uint32_t* particlesSplitCount = nullptr;
    uint32_t* particlesCount = nullptr;
    float* criterionValuesSplit = nullptr;
    float* criterionValuesMerge = nullptr;

    uint32_t* particlesIdsToMergeFirst = nullptr;
    uint32_t* particlesIdsToMergeSecond = nullptr;
    uint32_t* particlesMergeCount = nullptr;
    refinement::RefinementData::RemovalState* removalMarks = nullptr;
    uint32_t* prefixSums = nullptr;

    hipMalloc(&particlesIdsToSplit,
               static_cast<size_t>(static_cast<float>(maxParticleCount) * maxBatchSize) * sizeof(uint32_t));
    hipMalloc(&particlesSplitCount, sizeof(uint32_t));
    hipMalloc(&particlesCount, sizeof(uint32_t));
    hipMalloc(&criterionValuesSplit, maxParticleCount * sizeof(float));
    hipMalloc(&criterionValuesMerge, maxParticleCount * sizeof(float));
    hipMalloc(&particlesIds, maxParticleCount * sizeof(uint32_t));

    hipMalloc(&particlesIdsToMergeFirst,
               static_cast<size_t>(static_cast<float>(maxParticleCount) * maxBatchSize) * sizeof(uint32_t));
    hipMalloc(&particlesIdsToMergeSecond,
               static_cast<size_t>(static_cast<float>(maxParticleCount) * maxBatchSize) * sizeof(uint32_t));
    hipMalloc(&particlesMergeCount, sizeof(uint32_t));
    hipMalloc(&removalMarks, maxParticleCount * sizeof(uint32_t));
    hipMalloc(&prefixSums, maxParticleCount * sizeof(uint32_t));

    return {
        .split = {.criterionValues = {.data = criterionValuesSplit, .size = maxParticleCount},
                  .particlesIdsToSplit = {.data = particlesIdsToSplit,
                                          .size =
                                              static_cast<size_t>(static_cast<float>(maxParticleCount) * maxBatchSize)},
                  .particlesSplitCount = particlesSplitCount},
        .merge = {.criterionValues = {.data = criterionValuesMerge, .size = maxParticleCount},
                  .particlesIdsToMerge =
                      {Span {.data = particlesIdsToMergeFirst,
                             .size = static_cast<size_t>(static_cast<float>(maxParticleCount) * maxBatchSize)},
                       Span {.data = particlesIdsToMergeSecond,
                             .size = static_cast<size_t>(static_cast<float>(maxParticleCount) * maxBatchSize)}},
                  .removalFlags = {.data = removalMarks, .size = maxParticleCount},
                  .prefixSums = {.data = prefixSums, .size = maxParticleCount},
                  .particlesMergeCount = particlesMergeCount},
        .particlesIds = {.data = particlesIds, .size = maxParticleCount},
        .particlesCount = particlesCount
    };
}

void AdaptiveSphSimulation::update(const Parameters& parameters, float deltaTime)
{
    updateParameters(parameters);

    computeExternalForces(deltaTime);
    resetGrid();
    assignParticlesToCells();
    sortParticles();
    calculateCellStartAndEndIndices();
    computeDensities();
    computePressureForce(deltaTime);
    computeViscosityForce(deltaTime);
    integrateMotion(deltaTime);

    if (_frameCounter == _refinementParams.initialCooldown ||
        (_frameCounter > _refinementParams.initialCooldown && _frameCounter % _refinementParams.cooldown == 0))
    {
        performAdaptiveRefinement();
    }

    handleCollisions();

    hipDeviceSynchronize();
    _frameCounter++;
}

void AdaptiveSphSimulation::updateParticleCount()
{
    setParticleCount(fromGpu(_refinementData.particlesCount));
}

auto AdaptiveSphSimulation::getBlocksPerGridForParticles(uint32_t count) const -> dim3
{
    return {(count + getThreadsPerBlock() - 1) / getThreadsPerBlock()};
}

void AdaptiveSphSimulation::performAdaptiveRefinement()
{
    resetRefinementCounters();
    uint32_t currentCount = getParticlesCount();

    resetEnhancedMergeData(currentCount);
    // First perform merging and track how many particles were removed
    performMerging();
    // Calculate the new count after merging
    uint32_t postMergeCount = getParticlesCount();
    // Calculate how many particles were removed during merging
    _particlesRemovedInLastMerge = currentCount > postMergeCount ? currentCount - postMergeCount : 0;

    // Only split particles if we've removed some during merging
    if (_particlesRemovedInLastMerge > 0)
    {
        identifyAndSplitParticles();
    }

    // Update the final particle count
    setParticleCount(fromGpu(_refinementData.particlesCount));
}

void AdaptiveSphSimulation::resetRefinementCounters() const
{
    static constexpr uint32_t zero = 0;

    hipMemcpy(_refinementData.split.particlesSplitCount, &zero, sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(_refinementData.merge.particlesMergeCount, &zero, sizeof(uint32_t), hipMemcpyHostToDevice);
    // Reset criterion values arrays
    hipMemset(_refinementData.split.criterionValues.data,
               0,
               _refinementData.split.criterionValues.size * sizeof(float));
    hipMemset(_refinementData.merge.criterionValues.data,
               0,
               _refinementData.merge.criterionValues.size * sizeof(float));
    // Reset particle IDs array
    hipMemset(_refinementData.particlesIds.data, 0, _refinementData.particlesIds.size * sizeof(uint32_t));

    // Reset all removal flags and markers

    hipMemset(_refinementData.merge.removalFlags.data, 0, _refinementData.merge.removalFlags.size * sizeof(uint32_t));
    hipMemset(_refinementData.merge.prefixSums.data, 0, _refinementData.merge.prefixSums.size * sizeof(uint32_t));

    // Reset split particle IDs
    hipMemset(_refinementData.split.particlesIdsToSplit.data,
               0,
               _refinementData.split.particlesIdsToSplit.size * sizeof(uint32_t));
    // Reset merge particle IDs
    hipMemset(_refinementData.merge.particlesIdsToMerge.first.data,
               0,
               _refinementData.merge.particlesIdsToMerge.first.size * sizeof(uint32_t));
    hipMemset(_refinementData.merge.particlesIdsToMerge.second.data,
               0,
               _refinementData.merge.particlesIdsToMerge.second.size * sizeof(uint32_t));
}

void AdaptiveSphSimulation::identifyAndSplitParticles() const
{
    const float minMass = _refinementParams.minMassRatio * getParameters().baseParticleMass;

    if (_refinementParams.criterionType == "interface")
    {
        getCriterionValuesNoGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.split.criterionValues,
            refinement::interfaceCriterion::SplitCriterionGenerator(minMass, _refinementParams.interfaceParameters),
            getParameters());
    }
    else if (_refinementParams.criterionType == "vorticity")
    {
        getCriterionValuesWithGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.split.criterionValues,
            refinement::vorticity::SplitCriterionGenerator(minMass, _refinementParams.vorticity),
            getState().grid,
            getParameters());
    }
    else if (_refinementParams.criterionType == "curvature")
    {
        getCriterionValuesWithGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.split.criterionValues,
            refinement::curvature::SplitCriterionGenerator(minMass, _refinementParams.curvature),
            getState().grid,
            getParameters());
    }
    else
    {
        refinement::getCriterionValues<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.split.criterionValues,
            refinement::velocity::SplitCriterionGenerator(minMass, _refinementParams.velocity.split));
    }

    refinement::findTopParticlesToSplit(getParticles(), _refinementData, _refinementParams, thrust::greater<float> {});

    const auto particlesToSplitCount =
        std::min(fromGpu(_refinementData.split.particlesSplitCount), _particlesRemovedInLastMerge / 12);

    if (particlesToSplitCount == 0)
    {
        return;
    }

    refinement::splitParticles<<<getBlocksPerGridForParticles(particlesToSplitCount), getThreadsPerBlock()>>>(
        getParticles(),
        _refinementData,
        _refinementParams.splitting,
        _refinementParams.maxParticleCount);
}

void AdaptiveSphSimulation::identifyAndMergeParticles() const
{
    if (getParticlesCount() <= 1)
    {
        return;
    }

    uint32_t currentCount = fromGpu(_refinementData.particlesCount);

    thrust::fill(thrust::device,
                 _refinementData.merge.removalFlags.data,
                 _refinementData.merge.removalFlags.data + static_cast<size_t>(currentCount),
                 refinement::RefinementData::RemovalState::Default);

    const float maxMass = _refinementParams.maxMassRatio * getParameters().baseParticleMass;

    if (_refinementParams.criterionType == "interface")
    {
        getCriterionValuesNoGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.merge.criterionValues,
            refinement::interfaceCriterion::MergeCriterionGenerator(maxMass, _refinementParams.interfaceParameters),
            getParameters());
    }
    else if (_refinementParams.criterionType == "vorticity")
    {
        getCriterionValuesWithGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.merge.criterionValues,
            refinement::vorticity::MergeCriterionGenerator(maxMass, _refinementParams.vorticity),
            getState().grid,
            getParameters());
    }
    else if (_refinementParams.criterionType == "curvature")
    {
        getCriterionValuesWithGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.merge.criterionValues,
            refinement::curvature::MergeCriterionGenerator(maxMass, _refinementParams.curvature),
            getState().grid,
            getParameters());
    }
    else
    {
        refinement::getCriterionValues<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            _refinementData.merge.criterionValues,
            refinement::velocity::MergeCriterionGenerator(maxMass, _refinementParams.velocity.merge));
    }

    refinement::findTopParticlesToMerge(getParticles(), _refinementData, _refinementParams, thrust::less<float> {});

    const auto particlesToMergeCount = fromGpu(_refinementData.merge.particlesMergeCount);

    if (particlesToMergeCount == 0)
    {
        return;
    }

    refinement::getMergeCandidates<<<getBlocksPerGridForParticles(particlesToMergeCount), getThreadsPerBlock()>>>(
        getParticles(),
        _refinementData,
        getState().grid,
        getParameters());

    refinement::markPotentialMerges<<<getBlocksPerGridForParticles(particlesToMergeCount), getThreadsPerBlock()>>>(
        getParticles(),
        _refinementData);

    refinement::validateMergePairs<<<getBlocksPerGridForParticles(particlesToMergeCount), getThreadsPerBlock()>>>(
        _refinementData,
        getParticlesCount());

    refinement::mergeParticles<<<getBlocksPerGridForParticles(particlesToMergeCount), getThreadsPerBlock()>>>(
        getParticles(),
        _refinementData,
        getParameters());

    computePrefixSum();

    refinement::updateParticleCount<<<1, 1>>>(_refinementData, getParticlesCount());

    refinement::removeParticles<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
        getParticles(),
        _refinementData);
}

void AdaptiveSphSimulation::computePrefixSum() const
{
    thrust::transform(thrust::device,
                      _refinementData.merge.removalFlags.data,
                      _refinementData.merge.removalFlags.data + getParticlesCount(),
                      _refinementData.merge.prefixSums.data,
                      [] __device__(refinement::RefinementData::RemovalState flag) {
                          return (flag == refinement::RefinementData::RemovalState::Remove) ? 1 : 0;
                      });

    thrust::exclusive_scan(thrust::device,
                           _refinementData.merge.prefixSums.data,
                           _refinementData.merge.prefixSums.data + getParticlesCount(),
                           _refinementData.merge.prefixSums.data,
                           0);
}

void AdaptiveSphSimulation::performMerging()
{
    refinement::MergeConfiguration mergeConfig;
    mergeConfig.maxMassRatio = _refinementParams.maxMassRatio;
    mergeConfig.baseParticleMass = getParameters().baseParticleMass;
    mergeConfig.maxMassThreshold = mergeConfig.maxMassRatio * mergeConfig.baseParticleMass;

    const auto currentCount = getParticlesCount();

    hipMemset(_enhancedMergeData.eligibleCount, 0, sizeof(uint32_t));
    hipMemset(_enhancedMergeData.pairCount, 0, sizeof(uint32_t));

    hipMemset(_enhancedMergeData.states.data, 0, _enhancedMergeData.states.size * sizeof(refinement::MergeState));
    hipMemset(_enhancedMergeData.compactionMap.data, 0, _enhancedMergeData.compactionMap.size * sizeof(uint32_t));

    calculateMergeCriteria(_enhancedMergeData.criterionValues);

    refinement::identifyEligibleParticles<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
        getParticles(),
        _enhancedMergeData,
        _refinementParams.maxMassRatio * getParameters().baseParticleMass);
    hipDeviceSynchronize();
    const uint32_t eligibleCount = fromGpu(_enhancedMergeData.eligibleCount);
    if (eligibleCount == 0)
    {
        return;
    }
    proposePartners<<<getBlocksPerGridForParticles(eligibleCount), getThreadsPerBlock()>>>(getParticles(),
                                                                                           _enhancedMergeData,
                                                                                           getState().grid,
                                                                                           getParameters(),
                                                                                           mergeConfig);
    resolveProposals<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(getParticles(),
                                                                                              _enhancedMergeData);

    refinement::createMergePairs<<<getBlocksPerGridForParticles(eligibleCount), getThreadsPerBlock()>>>(
        _enhancedMergeData);
    hipDeviceSynchronize();
    const uint32_t pairCount = fromGpu(_enhancedMergeData.pairCount);
    if (pairCount == 0)
    {
        return;
    }
    refinement::executeMerges<<<getBlocksPerGridForParticles(pairCount), getThreadsPerBlock()>>>(getParticles(),
                                                                                                 _enhancedMergeData,
                                                                                                 getParameters());
    refinement::buildCompactionMap<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
        _enhancedMergeData,
        currentCount);
    thrust::exclusive_scan(thrust::device,
                           _enhancedMergeData.compactionMap.data,
                           _enhancedMergeData.compactionMap.data + currentCount,
                           _enhancedMergeData.compactionMap.data);

    refinement::compactParticles<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
        getParticles(),
        _enhancedMergeData,
        currentCount);
    const uint32_t removedCount = fromGpu(&_enhancedMergeData.compactionMap.data[currentCount - 1]);
    const uint32_t newCount = currentCount - removedCount;

    setParticleCount(newCount);
    hipMemcpy(_refinementData.particlesCount, &newCount, sizeof(uint32_t), hipMemcpyHostToDevice);
}

refinement::EnhancedMergeData AdaptiveSphSimulation::allocateEnhancedMergeData(uint32_t maxParticleCount)
{
    refinement::EnhancedMergeData data;

    hipMalloc(&data.criterionValues.data, maxParticleCount * sizeof(float));
    data.criterionValues.size = maxParticleCount;

    hipMalloc(&data.eligibleParticles.data, maxParticleCount * sizeof(uint32_t));
    data.eligibleParticles.size = maxParticleCount;

    hipMalloc(&data.eligibleCount, sizeof(uint32_t));

    hipMalloc(&data.states.data, maxParticleCount * sizeof(refinement::MergeState));
    data.states.size = maxParticleCount;

    const size_t maxPairs = maxParticleCount / 2;
    hipMalloc(&data.pairs.data, maxPairs * sizeof(refinement::MergePair));
    data.pairs.size = maxPairs;

    hipMalloc(&data.pairCount, sizeof(uint32_t));

    hipMalloc(&data.compactionMap.data, maxParticleCount * sizeof(uint32_t));
    data.compactionMap.size = maxParticleCount;

    hipMalloc(&data.newParticleCount, sizeof(uint32_t));

    return data;
}

void AdaptiveSphSimulation::calculateMergeCriteria(Span<float> criterionValues) const
{
    const float maxMass = _refinementParams.maxMassRatio * getParameters().baseParticleMass;

    if (_refinementParams.criterionType == "interface")
    {
        getCriterionValuesNoGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            criterionValues,
            refinement::interfaceCriterion::MergeCriterionGenerator(maxMass, _refinementParams.interfaceParameters),
            getParameters());
    }
    else if (_refinementParams.criterionType == "vorticity")
    {
        getCriterionValuesWithGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            criterionValues,
            refinement::vorticity::MergeCriterionGenerator(maxMass, _refinementParams.vorticity),
            getState().grid,
            getParameters());
    }
    else if (_refinementParams.criterionType == "curvature")
    {
        getCriterionValuesWithGrid<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            criterionValues,
            refinement::curvature::MergeCriterionGenerator(maxMass, _refinementParams.curvature),
            getState().grid,
            getParameters());
    }
    else
    {
        refinement::getCriterionValues<<<SphSimulation::getBlocksPerGridForParticles(), getThreadsPerBlock()>>>(
            getParticles(),
            criterionValues,
            refinement::velocity::MergeCriterionGenerator(maxMass, _refinementParams.velocity.merge));
    }
}

void AdaptiveSphSimulation::freeEnhancedMergeData(const refinement::EnhancedMergeData& data)
{
    hipFree(data.criterionValues.data);
    hipFree(data.eligibleParticles.data);
    hipFree(data.eligibleCount);
    hipFree(data.states.data);
    hipFree(data.pairs.data);
    hipFree(data.pairCount);
    hipFree(data.compactionMap.data);
    hipFree(data.newParticleCount);
}

void AdaptiveSphSimulation::resetEnhancedMergeData(uint32_t currentParticleCount) const
{
    // Reset counters
    hipMemset(_enhancedMergeData.eligibleCount, 0, sizeof(uint32_t));
    hipMemset(_enhancedMergeData.pairCount, 0, sizeof(uint32_t));
    hipMemset(_enhancedMergeData.newParticleCount, 0, sizeof(uint32_t));
    // Clear state arrays    hipMemset(_enhancedMergeData.states.data, 0,_enhancedMergeData.states.size * sizeof(refinement::MergeState));
    // Clear criterion values
    hipMemset(_enhancedMergeData.criterionValues.data, 0, _enhancedMergeData.criterionValues.size * sizeof(float));
    // Clear compaction map up to current particle count
    hipMemset(_enhancedMergeData.compactionMap.data, 0, currentParticleCount * sizeof(uint32_t));
    // Clear eligible particles array
    hipMemset(_enhancedMergeData.eligibleParticles.data,
               0,
               _enhancedMergeData.eligibleParticles.size * sizeof(uint32_t));

    // Clear pairs array
    hipMemset(_enhancedMergeData.pairs.data, 0, _enhancedMergeData.pairs.size * sizeof(refinement::MergePair));
}
}
