#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include <cstddef>
#include <memory>

#include "ImportedParticleMemory.cuh"
#include "cuda/ImportedMemory.cuh"
#include "cuda/Simulation.cuh"

namespace sph::cuda
{

ImportedParticleMemory::~ImportedParticleMemory()
{
    hipFree(_particles);
    hipDestroyExternalMemory(_externalMemory);
}
#if defined(WIN32)
ImportedParticleMemory::ImportedParticleMemory(void* handle, size_t size)
    : _size {size}
{
    hipExternalMemoryHandleDesc handleDesc {};
    handleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    handleDesc.size = size;
    handleDesc.handle.win32.handle = handle;

    hipImportExternalMemory(&_externalMemory, &handleDesc);

    hipExternalMemoryBufferDesc bufferDesc {};
    bufferDesc.size = size;
    bufferDesc.offset = 0;

    hipExternalMemoryGetMappedBuffer(reinterpret_cast<void**>(&_particles), _externalMemory, &bufferDesc);
}
#else
ImportedParticleMemory::ImportedParticleMemory(int handle, size_t size)
    : _size {size}
{
    hipExternalMemoryHandleDesc handleDesc {};
    handleDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
    handleDesc.size = size;
    handleDesc.handle.fd = handle;

    hipImportExternalMemory(&_externalMemory, &handleDesc);

    hipExternalMemoryBufferDesc bufferDesc {};
    bufferDesc.size = size;
    bufferDesc.offset = 0;

    hipExternalMemoryGetMappedBuffer(reinterpret_cast<void**>(&_particles), _externalMemory, &bufferDesc);
}
#endif

auto ImportedParticleMemory::getParticles() const -> ParticleData*
{
    return _particles;
}

auto ImportedParticleMemory::getSize() const -> size_t
{
    return _size;
}

auto ImportedParticleMemory::getMaxParticleCount() const -> size_t
{
    return _size / sizeof(ParticleData);
}

#if defined(WIN32)
auto importBuffer(void* handle, size_t size) -> std::unique_ptr<ImportedMemory>
{
    return std::make_unique<ImportedParticleMemory>(handle, size);
}
#else
auto importBuffer(int handle, size_t size) -> std::unique_ptr<ImportedMemory>
{
    return std::make_unique<ImportedParticleMemory>(handle, size);
}
#endif

}
