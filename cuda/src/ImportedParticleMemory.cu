#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include <cstddef>
#include <cuda/ImportedMemory.cuh>
#include <memory>

#include "ImportedParticleMemory.cuh"

namespace sph::cuda
{

ImportedParticleMemory::~ImportedParticleMemory()
{
    hipFree(_data);
    hipDestroyExternalMemory(_externalMemory);
}
#if defined(WIN32)
ImportedParticleMemory::ImportedParticleMemory(void* handle, size_t size)
    : _size {size}
{
    hipExternalMemoryHandleDesc handleDesc {};
    handleDesc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    handleDesc.size = size;
    handleDesc.handle.win32.handle = handle;

    hipImportExternalMemory(&_externalMemory, &handleDesc);

    hipExternalMemoryBufferDesc bufferDesc {};
    bufferDesc.size = size;
    bufferDesc.offset = 0;

    hipExternalMemoryGetMappedBuffer(&_data, _externalMemory, &bufferDesc);
}
#else
ImportedParticleMemory::ImportedParticleMemory(int handle, size_t size)
    : _size {size}
{
    hipExternalMemoryHandleDesc handleDesc {};
    handleDesc.type = hipExternalMemoryHandleTypeOpaqueFd;
    handleDesc.size = size;
    handleDesc.handle.fd = handle;

    hipImportExternalMemory(&_externalMemory, &handleDesc);

    hipExternalMemoryBufferDesc bufferDesc {};
    bufferDesc.size = size;
    bufferDesc.offset = 0;

    hipExternalMemoryGetMappedBuffer(&_data, _externalMemory, &bufferDesc);
}
#endif

#if defined(WIN32)
auto importBuffer(void* handle, size_t size) -> std::unique_ptr<ImportedMemory>
{
    return std::make_unique<ImportedParticleMemory>(handle, size);
}
#else
auto importBuffer(int handle, size_t size) -> std::unique_ptr<ImportedMemory>
{
    return std::make_unique<ImportedParticleMemory>(handle, size);
}
#endif

}
