#include "hip/hip_runtime.h"
#include <cstdint>
#include <cuda/Simulation.cuh>
#include <glm/ext/vector_uint3.hpp>

#include "Algorithm.cuh"
#include "Span.cuh"
#include "SphSimulation.cuh"
#include "common/Iteration.hpp"
#include "common/Utils.cuh"
#include "device/Kernel.cuh"
#include "glm/geometric.hpp"

namespace sph::cuda::kernel
{

__device__ void handleCollision(ParticlesData particles, uint32_t id, const Simulation::Parameters& simulationData);

__global__ void handleCollisions(ParticlesData particles, Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < particles.particleCount)
    {
        handleCollision(particles, idx, simulationData);
    }
}

__global__ void resetGrid(SphSimulation::Grid grid)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < grid.cellStartIndices.size)
    {
        grid.cellStartIndices.data[idx] = -1;
        grid.cellEndIndices.data[idx] = -1;
    }
}

__global__ void assignParticlesToCells(ParticlesData particles,
                                       SphSimulation::State state,
                                       Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < particles.particleCount)
    {
        state.grid.particleArrayIndices.data[idx] = idx;
        const auto cellPosition = calculateCellIndex(particles.positions[idx], simulationData, state.grid);
        const auto cellIndex = flattenCellIndex(cellPosition, state.grid.gridSize);
        state.grid.particleGridIndices.data[idx] = cellIndex;
    }
}

__global__ void calculateCellStartAndEndIndices(SphSimulation::Grid grid, uint32_t particleCount)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particleCount)
    {
        return;
    }

    const auto cellIdx = grid.particleGridIndices.data[idx];

    if (idx == 0 || grid.particleGridIndices.data[idx - 1] != cellIdx)
    {
        grid.cellStartIndices.data[cellIdx] = idx;
    }
    if (idx == grid.particleArrayIndices.size - 1 || grid.particleGridIndices.data[idx + 1] != cellIdx)
    {
        grid.cellEndIndices.data[cellIdx] = idx;
    }
}

__global__ void computeDensities(ParticlesData particles,
                                 SphSimulation::State state,
                                 Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    const auto position = particles.predictedPositions[idx];
    const auto radiusSquared = particles.smoothingRadiuses[idx] * particles.smoothingRadiuses[idx];

    auto density = 0.F;
    auto nearDensity = 0.F;

    forEachNeighbour(position, simulationData, state.grid, [&](const auto neighbourIdx) {
        const auto neighbourPosition = particles.predictedPositions[neighbourIdx];
        const auto offsetToNeighbour = neighbourPosition - position;
        const auto distanceSquared = glm::dot(offsetToNeighbour, offsetToNeighbour);

        if (distanceSquared > radiusSquared)
        {
            return;
        }
        const auto distance = glm::sqrt(distanceSquared);
        const auto neighbourMass = particles.masses[neighbourIdx];

        density += neighbourMass * device::densityKernel(distance, particles.smoothingRadiuses[idx]);
        nearDensity += neighbourMass * device::nearDensityKernel(distance, particles.smoothingRadiuses[idx]);
    });

    particles.densities[idx] = density;
    particles.nearDensities[idx] = nearDensity;
}

__global__ void computePressureForce(ParticlesData particles,
                                     SphSimulation::State state,
                                     Simulation::Parameters simulationData,
                                     float dt)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }
    const auto position = particles.predictedPositions[idx];
    const auto density = particles.densities[idx];
    const auto nearDensity = particles.nearDensities[idx];
    const auto pressure = (density - simulationData.restDensity) * simulationData.pressureConstant;
    const auto nearPressure = nearDensity * simulationData.nearPressureConstant;

    auto pressureForce = glm::vec4 {};
    const auto radiusSquared = particles.smoothingRadiuses[idx] * particles.smoothingRadiuses[idx];

    forEachNeighbour(position, simulationData, state.grid, [&](const auto neighbourIdx) {
        if (neighbourIdx == idx)
        {
            return;
        }

        const auto neighbourPosition = particles.predictedPositions[neighbourIdx];
        const auto offsetToNeighbour = neighbourPosition - position;
        const auto distanceSquared = glm::dot(offsetToNeighbour, offsetToNeighbour);

        if (distanceSquared > radiusSquared)
        {
            return;
        }
        const auto densityNeighbour = particles.densities[neighbourIdx];
        const auto nearDensityNeighbour = particles.nearDensities[neighbourIdx];
        const auto pressureNeighbour =
            (densityNeighbour - simulationData.restDensity) * simulationData.pressureConstant;
        const auto nearPressureNeighbour = nearDensityNeighbour * simulationData.nearPressureConstant;

        const auto sharedPressure = (pressure + pressureNeighbour) / 2.F;
        const auto sharedNearPressure = (nearPressure + nearPressureNeighbour) / 2.F;

        const auto distance = glm::sqrt(distanceSquared);
        const auto direction = distance > 0.F ? offsetToNeighbour / distance : glm::vec4(0.F, 1.F, 0.F, 0.F);

        const auto neighbourMass = particles.masses[neighbourIdx];

        pressureForce += neighbourMass * direction *
                         device::densityDerivativeKernel(distance, particles.smoothingRadiuses[idx]) * sharedPressure /
                         densityNeighbour;
        pressureForce += neighbourMass * direction *
                         device::nearDensityDerivativeKernel(distance, particles.smoothingRadiuses[idx]) *
                         sharedNearPressure / nearDensityNeighbour;
    });

    const auto particleMass = particles.masses[idx];
    const auto acceleration = pressureForce / particleMass;

    particles.velocities[idx] += acceleration * dt;
}

__global__ void computeViscosityForce(ParticlesData particles,
                                      SphSimulation::State state,
                                      Simulation::Parameters simulationData,
                                      float dt)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }
    const auto position = particles.predictedPositions[idx];
    const auto velocity = particles.velocities[idx];

    auto viscosityForce = glm::vec4 {};

    const auto radiusSquared = particles.smoothingRadiuses[idx] * particles.smoothingRadiuses[idx];

    forEachNeighbour(position, simulationData, state.grid, [&](const auto neighbourIdx) {
        if (neighbourIdx == idx)
        {
            return;
        }

        const auto neighbourPosition = particles.predictedPositions[neighbourIdx];
        const auto offsetToNeighbour = neighbourPosition - position;
        const auto distanceSquared = glm::dot(offsetToNeighbour, offsetToNeighbour);

        if (distanceSquared > radiusSquared)
        {
            return;
        }

        const auto distance = glm::sqrt(distanceSquared);
        const auto neighbourVelocity = particles.velocities[neighbourIdx];
        const auto neighbourMass = particles.masses[neighbourIdx];
        const auto neighbourDensity = particles.densities[neighbourIdx];

        viscosityForce += neighbourMass * (neighbourVelocity - velocity) / neighbourDensity *
                          device::viscosityLaplacianKernel(distance, particles.smoothingRadiuses[idx]);
    });

    const auto particleMass = particles.masses[idx];
    const auto acceleration = simulationData.viscosityConstant * viscosityForce / particleMass;
    particles.velocities[idx] += acceleration * dt / particleMass;
}

__global__ void integrateMotion(ParticlesData particles, Simulation::Parameters simulationData, float dt)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    const auto velocityMagnitude = glm::length(particles.velocities[idx]);
    if (velocityMagnitude > simulationData.maxVelocity)
    {
        particles.velocities[idx] *= simulationData.maxVelocity / velocityMagnitude;
    }
    particles.positions[idx] += particles.velocities[idx] * dt;
}

__device__ void handleCollision(ParticlesData particles, uint32_t id, const Simulation::Parameters& simulationData)
{
    for (int i = 0; i < 3; i++)
    {
        const auto minBoundary = simulationData.domain.min[i] + particles.radiuses[id];
        const auto maxBoundary = simulationData.domain.max[i] - particles.radiuses[id];

        if (particles.positions[id][i] < minBoundary)
        {
            particles.positions[id][i] = minBoundary;
            particles.velocities[id][i] = -particles.velocities[id][i] * simulationData.restitution;
        }

        if (particles.positions[id][i] > maxBoundary)
        {
            particles.positions[id][i] = maxBoundary;
            particles.velocities[id][i] = -particles.velocities[id][i] * simulationData.restitution;
        }
    }
}

__global__ void computeExternalForces(ParticlesData particles, Simulation::Parameters simulationData, float deltaTime)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    particles.velocities[idx] += glm::vec4 {simulationData.gravity, 0.F} * deltaTime;
    particles.predictedPositions[idx] = particles.positions[idx] + particles.velocities[idx] * 1.F / 120.F;
}
}
