#include "CudaTimer.cuh"

namespace sph::benchmark
{

CudaTimer::CudaTimer()
{
    if (hipEventCreate(&_startEvent) == hipSuccess && hipEventCreate(&_stopEvent) == hipSuccess)
    {
        _initialized = true;
    }
    else
    {
        _initialized = false;
    }
}

CudaTimer::~CudaTimer()
{
    if (_initialized)
    {
        hipEventDestroy(_startEvent);
        hipEventDestroy(_stopEvent);
    }
}

void CudaTimer::start()
{
    if (!_initialized)
    {
        return;
    }
    if (_isRunning)
    {
        return;
    }
    hipEventRecord(_startEvent);
    _isRunning = true;
}

auto CudaTimer::stop() -> float
{
    if (!_initialized || !_isRunning)
    {
        return 0.0F;
    }
    hipEventRecord(_stopEvent);
    hipEventSynchronize(_stopEvent);

    float elapsedTime = 0.0F;
    hipEventElapsedTime(&elapsedTime, _startEvent, _stopEvent);

    _isRunning = false;
    return elapsedTime;  // Returns time in milliseconds
}

}
