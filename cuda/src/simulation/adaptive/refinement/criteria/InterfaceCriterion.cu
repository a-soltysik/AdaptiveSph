#include "hip/hip_runtime.h"
#include <cstdint>
#include <glm/common.hpp>
#include <glm/ext/vector_float3.hpp>
#include <glm/ext/vector_float4.hpp>
#include <glm/glm.hpp>

#include "../../../SphSimulation.cuh"
#include "InterfaceCriterion.cuh"
#include "cuda/Simulation.cuh"
#include "glm/gtx/component_wise.hpp"

namespace sph::cuda::refinement::interfaceCriterion
{

__device__ auto calculateMinSurfaceDistance(const glm::vec4& position,
                                            const Simulation::Parameters::Domain& domain,
                                            const Simulation::Parameters& simulationData) -> glm::vec3
{
    const auto distToMinX = position.x - domain.min.x;
    const auto distToMaxX = domain.max.x - position.x;
    const auto distToMinY = position.y - domain.min.y;
    const auto distToMaxY = domain.max.y - position.y;
    const auto distToMinZ = position.z - domain.min.z;
    const auto distToMaxZ = domain.max.z - position.z;

    const auto minDistX = glm::min(distToMinX, distToMaxX);
    const auto minDistY = glm::min(distToMinY, distToMaxY);
    const auto minDistZ = glm::min(distToMinZ, distToMaxZ);

    if (simulationData.testCase == Simulation::Parameters::TestCase::PoiseuilleFlow)
    {
        return {FLT_MAX, glm::max(0.0F, minDistY), glm::max(0.0F, minDistZ)};
    }

    return {glm::max(0.0F, minDistX), glm::max(0.0F, minDistY), glm::max(0.0F, minDistZ)};
}

__device__ auto SplitCriterionGenerator::operator()(ParticlesData particles,
                                                    uint32_t id,
                                                    const SphSimulation::Grid& grid,
                                                    const Simulation::Parameters& simulationData) const -> float
{
    if (particles.masses[id] < _minimalMass)
    {
        return -1.0F;
    }

    const auto position = particles.positions[id];
    const auto minDistances = calculateMinSurfaceDistance(position, simulationData.domain, simulationData);
    const auto domainSize = simulationData.domain.max - simulationData.domain.min;
    const auto splitThresholds = _interface.split.distanceRatioThreshold * domainSize;
    const auto normalizedDistances = minDistances / splitThresholds;
    const auto minNormalizedDistance = glm::compMin(normalizedDistances);

    return 1.0F - minNormalizedDistance;
}

__device__ auto MergeCriterionGenerator::operator()(ParticlesData particles,
                                                    uint32_t id,
                                                    const SphSimulation::Grid& grid,
                                                    const Simulation::Parameters& simulationData) const -> float
{
    if (particles.masses[id] > _maximalMass)
    {
        return -1.0F;
    }

    const auto position = particles.positions[id];
    const auto minDistances = calculateMinSurfaceDistance(position, simulationData.domain, simulationData);
    const auto domainSize = simulationData.domain.max - simulationData.domain.min;
    const auto mergeThresholds = _interface.merge.distanceRatioThreshold * domainSize;
    const auto normalizedDistances = minDistances / mergeThresholds;
    const auto minNormalizedDistance = glm::compMin(normalizedDistances);

    return minNormalizedDistance - 1.0F;
}

}
