#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdint>
#include <cuda/Simulation.cuh>
#include <glm/exponential.hpp>
#include <glm/ext/vector_float3.hpp>
#include <glm/ext/vector_float4.hpp>

#include "Algorithm.cuh"
#include "glm/ext/scalar_constants.hpp"
#include "glm/geometric.hpp"
#include "kernels/Kernel.cuh"
#include "simulation/adaptive/SphSimulation.cuh"
#include "utils/Iteration.cuh"
#include "utils/Utils.cuh"

namespace sph::cuda::kernel
{

__device__ void handleCollision(ParticlesData particles, uint32_t id, const Simulation::Parameters& simulationData);
__device__ void handleLidDrivenCavityCollision(ParticlesData particles,
                                               uint32_t id,
                                               const Simulation::Parameters& simulationData);
__device__ void handlePoiseuilleFlowCollision(ParticlesData particles,
                                              uint32_t id,
                                              const Simulation::Parameters& simulationData);
__device__ void handleTaylorGreenVortexCollision(ParticlesData particles,
                                                 uint32_t id,
                                                 const Simulation::Parameters& simulationData);
__device__ void handleStandardCollision(ParticlesData particles,
                                        uint32_t id,
                                        const Simulation::Parameters& simulationData);
__device__ void handleNoSlipBoundaries(ParticlesData particles,
                                       uint32_t id,
                                       const Simulation::Parameters& simulationData,
                                       int axis);
__device__ void handleStandardBoundariesForAxis(ParticlesData particles,
                                                uint32_t id,
                                                const Simulation::Parameters& simulationData,
                                                int axis);
__device__ void handlePeriodicBoundariesForAxis(ParticlesData particles,
                                                uint32_t id,
                                                const Simulation::Parameters& simulationData,
                                                int axis);

__global__ void handleCollisions(ParticlesData particles, Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < particles.particleCount)
    {
        handleCollision(particles, idx, simulationData);
    }
}

__global__ void resetGrid(SphSimulation::Grid grid)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < grid.cellStartIndices.size())
    {
        grid.cellStartIndices[idx] = -1;
        grid.cellEndIndices[idx] = -1;
    }
}

__global__ void assignParticlesToCells(ParticlesData particles,
                                       SphSimulation::State state,
                                       Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx < particles.particleCount)
    {
        state.grid.particleArrayIndices[idx] = idx;
        const auto cellPosition = calculateCellIndex(particles.predictedPositions[idx], simulationData, state.grid);
        const auto cellIndex = flattenCellIndex(cellPosition, state.grid.gridSize);
        state.grid.particleGridIndices[idx] = cellIndex;
    }
}

__global__ void calculateCellStartAndEndIndices(SphSimulation::Grid grid, uint32_t particleCount)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particleCount)
    {
        return;
    }

    const auto cellIdx = grid.particleGridIndices[idx];

    if (idx == 0 || grid.particleGridIndices[idx - 1] != cellIdx)
    {
        grid.cellStartIndices[cellIdx] = idx;
    }
    if (idx == grid.particleArrayIndices.size() - 1 || grid.particleGridIndices[idx + 1] != cellIdx)
    {
        grid.cellEndIndices[cellIdx] = idx;
    }
}

__global__ void computeDensities(ParticlesData particles,
                                 SphSimulation::State state,
                                 Simulation::Parameters simulationData)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    const auto position = particles.predictedPositions[idx];
    auto density = 0.F;
    auto nearDensity = 0.F;

    forEachNeighbour(position,
                     particles,
                     simulationData,
                     state.grid,
                     [&](const auto neighbourIdx, const glm::vec4& adjustedPos) {
                         //if (neighbourIdx == idx)
                         //{
                         //    return;
                         //}
                         const auto offsetToNeighbour = adjustedPos - position;
                         const auto distanceSquared = glm::dot(offsetToNeighbour, offsetToNeighbour);

                         const auto neighbourSmoothingRadius = particles.smoothingRadiuses[neighbourIdx];
                         const auto radiusSquared = 4 * neighbourSmoothingRadius * neighbourSmoothingRadius;

                         if (distanceSquared > radiusSquared)
                         {
                             return;
                         }

                         const auto distance = glm::sqrt(distanceSquared);
                         const auto neighbourMass = particles.masses[neighbourIdx];

                         density += neighbourMass * device::densityKernel(distance, neighbourSmoothingRadius);
                         nearDensity += neighbourMass * device::nearDensityKernel(distance, neighbourSmoothingRadius);
                     });

    particles.densities[idx] = density;
    particles.nearDensities[idx] = nearDensity;
}

__global__ void computePressureForce(ParticlesData particles,
                                     SphSimulation::State state,
                                     Simulation::Parameters simulationData,
                                     float dt)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }
    const auto position = particles.predictedPositions[idx];
    const auto density = particles.densities[idx];
    const auto nearDensity = particles.nearDensities[idx];
    const auto pressure = (density - simulationData.restDensity) * simulationData.pressureConstant;
    const auto nearPressure = nearDensity * simulationData.nearPressureConstant;

    auto pressureForce = glm::vec4 {};

    forEachNeighbour(
        position,
        particles,
        simulationData,
        state.grid,
        [&](const auto neighbourIdx, const glm::vec4& adjustedPos) {
            if (neighbourIdx == idx)
            {
                return;
            }

            const auto offsetToNeighbour = adjustedPos - position;
            const auto distanceSquared = glm::dot(offsetToNeighbour, offsetToNeighbour);
            const auto neighbourSmoothingRadius = particles.smoothingRadiuses[neighbourIdx];
            const auto radiusSquared = 4 * neighbourSmoothingRadius * neighbourSmoothingRadius;

            if (distanceSquared > radiusSquared)
            {
                return;
            }
            const auto densityNeighbour = particles.densities[neighbourIdx];
            const auto nearDensityNeighbour = particles.nearDensities[neighbourIdx];
            const auto pressureNeighbour =
                (densityNeighbour - simulationData.restDensity) * simulationData.pressureConstant;
            const auto nearPressureNeighbour = nearDensityNeighbour * simulationData.nearPressureConstant;

            const auto sharedPressure = (pressure + pressureNeighbour) / 2.F;
            const auto sharedNearPressure = (nearPressure + nearPressureNeighbour) / 2.F;

            const auto distance = glm::sqrt(distanceSquared);
            const auto direction = distance > 0.F ? offsetToNeighbour / distance : glm::vec4(0.F, 1.F, 0.F, 0.F);

            const auto neighbourMass = particles.masses[neighbourIdx];

            pressureForce += neighbourMass * direction *
                             device::densityDerivativeKernel(distance, neighbourSmoothingRadius) * sharedPressure /
                             densityNeighbour;
            pressureForce += neighbourMass * direction *
                             device::nearDensityDerivativeKernel(distance, neighbourSmoothingRadius) *
                             sharedNearPressure / nearDensityNeighbour;
            //pressureForce +=
            //    direction * neighbourMass *
            //    (pressure / (density * density) + pressureNeighbour / (densityNeighbour * densityNeighbour)) *
            //    device::densityDerivativeKernel(distance, neighbourSmoothingRadius);
            //
            //pressureForce += direction * neighbourMass *
            //                 (nearPressure / (nearDensity * nearDensity) +
            //                  nearPressureNeighbour / (nearDensityNeighbour * nearDensityNeighbour)) *
            //                 device::nearDensityDerivativeKernel(distance, neighbourSmoothingRadius);
        });

    const auto particleMass = particles.masses[idx];
    const auto acceleration = pressureForce / particleMass;

    particles.velocities[idx] += acceleration * dt;
}

__global__ void computeViscosityForce(ParticlesData particles,
                                      SphSimulation::State state,
                                      Simulation::Parameters simulationData,
                                      float dt)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    const auto position = particles.predictedPositions[idx];
    const auto velocity = particles.velocities[idx];

    auto viscosityForce = glm::vec4 {};

    forEachNeighbour(position,
                     particles,
                     simulationData,
                     state.grid,
                     [&](const auto neighbourIdx, const glm::vec4& adjustedPos) {
                         if (neighbourIdx == idx)
                         {
                             return;
                         }

                         const auto offsetToNeighbour = adjustedPos - position;
                         const auto distanceSquared = glm::dot(offsetToNeighbour, offsetToNeighbour);
                         const auto smoothingRadius = particles.smoothingRadiuses[neighbourIdx];
                         const auto radiusSquared = smoothingRadius * smoothingRadius;

                         if (distanceSquared > radiusSquared)
                         {
                             return;
                         }

                         const auto distance = glm::sqrt(distanceSquared);
                         const auto neighbourVelocity = particles.velocities[neighbourIdx];
                         const auto neighbourMass = particles.masses[neighbourIdx];
                         const auto neighbourDensity = particles.densities[neighbourIdx];

                         viscosityForce += neighbourMass * (neighbourVelocity - velocity) / neighbourDensity *
                                           device::viscosityLaplacianKernel(distance, smoothingRadius);
                     });

    const auto particleMass = particles.masses[idx];
    const auto acceleration = simulationData.viscosityConstant * viscosityForce / particleMass;
    particles.velocities[idx] += acceleration * dt;
}

__global__ void integrateMotion(ParticlesData particles, Simulation::Parameters simulationData, float dt)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    const auto velocityMagnitude = glm::length(particles.velocities[idx]);
    if (velocityMagnitude > simulationData.maxVelocity)
    {
        particles.velocities[idx] *= simulationData.maxVelocity / velocityMagnitude;
    }
    particles.positions[idx] += particles.velocities[idx] * dt;
}

__device__ void handleCollision(ParticlesData particles, uint32_t id, const Simulation::Parameters& simulationData)
{
    switch (simulationData.testCase)
    {
    case Simulation::Parameters::TestCase::LidDrivenCavity:
        handleLidDrivenCavityCollision(particles, id, simulationData);
        break;
    case Simulation::Parameters::TestCase::PoiseuilleFlow:
        handlePoiseuilleFlowCollision(particles, id, simulationData);
        break;
    case Simulation::Parameters::TestCase::TaylorGreenVortex:
        handleTaylorGreenVortexCollision(particles, id, simulationData);
        break;
    default:
        handleStandardCollision(particles, id, simulationData);
        break;
    }
}

__device__ void handleLidDrivenCavityCollision(ParticlesData particles,
                                               uint32_t id,
                                               const Simulation::Parameters& simulationData)
{
    for (int i = 0; i < 3; i++)
    {
        const auto minBoundary = simulationData.domain.min[i] + particles.radiuses[id];
        const auto maxBoundary = simulationData.domain.max[i] - particles.radiuses[id];

        if (particles.positions[id][i] < minBoundary)
        {
            particles.positions[id][i] = minBoundary;
            if (i == 1)
            {
                // Top wall moves at lid velocity
                particles.velocities[id] = glm::vec4(simulationData.lidVelocity, 0.5F, 0.0F, 0.0F);
            }
            else
            {
                particles.velocities[id][i] = -particles.velocities[id][i] * simulationData.restitution;
            }
        }

        if (particles.positions[id][i] > maxBoundary)
        {
            particles.positions[id][i] = maxBoundary;
            particles.velocities[id][i] = -particles.velocities[id][i] * simulationData.restitution;
        }
    }
}

__device__ void handlePoiseuilleFlowCollision(ParticlesData particles,
                                              uint32_t id,
                                              const Simulation::Parameters& simulationData)
{
    // Handle y-axis (channel height) - no-slip boundaries
    handleNoSlipBoundaries(particles, id, simulationData, 1);

    // Handle z-axis (channel width) - standard boundaries
    handleStandardBoundariesForAxis(particles, id, simulationData, 2);

    // Handle x-axis (flow direction) - periodic boundaries
    handlePeriodicBoundariesForAxis(particles, id, simulationData, 0);
}

__device__ void handleTaylorGreenVortexCollision(ParticlesData particles,
                                                 uint32_t id,
                                                 const Simulation::Parameters& simulationData)
{
    const auto domainMin = simulationData.domain.min;
    const auto domainMax = simulationData.domain.max;
    const auto domainSize = domainMax - domainMin;

    // Handle periodic boundaries in all directions
    for (int i = 0; i < 3; i++)
    {
        if (particles.positions[id][i] < domainMin[i])
        {
            particles.positions[id][i] += domainSize[i];
            particles.predictedPositions[id][i] += domainSize[i];
        }
        else if (particles.positions[id][i] >= domainMax[i])
        {
            particles.positions[id][i] -= domainSize[i];
            particles.predictedPositions[id][i] -= domainSize[i];
        }
    }
}

__device__ void handleStandardCollision(ParticlesData particles,
                                        uint32_t id,
                                        const Simulation::Parameters& simulationData)
{
    for (int i = 0; i < 3; i++)
    {
        handleStandardBoundariesForAxis(particles, id, simulationData, i);
    }
}

__device__ void handleNoSlipBoundaries(ParticlesData particles,
                                       uint32_t id,
                                       const Simulation::Parameters& simulationData,
                                       int axis)
{
    const auto minBoundary = simulationData.domain.min[axis] + particles.radiuses[id];
    const auto maxBoundary = simulationData.domain.max[axis] - particles.radiuses[id];

    if (particles.positions[id][axis] < minBoundary)
    {
        particles.positions[id][axis] = minBoundary;
        particles.velocities[id] = -particles.velocities[id] * 0.2F;
    }

    if (particles.positions[id][axis] > maxBoundary)
    {
        particles.positions[id][axis] = maxBoundary;
        particles.velocities[id] = -particles.velocities[id] * 0.2F;
    };
}

__device__ void handleStandardBoundariesForAxis(ParticlesData particles,
                                                uint32_t id,
                                                const Simulation::Parameters& simulationData,
                                                int axis)
{
    const auto minBoundary = simulationData.domain.min[axis] + particles.radiuses[id];
    const auto maxBoundary = simulationData.domain.max[axis] - particles.radiuses[id];

    if (particles.positions[id][axis] < minBoundary)
    {
        particles.positions[id][axis] = minBoundary;
        particles.velocities[id][axis] = -particles.velocities[id][axis] * simulationData.restitution;
    }

    if (particles.positions[id][axis] > maxBoundary)
    {
        particles.positions[id][axis] = maxBoundary;
        particles.velocities[id][axis] = -particles.velocities[id][axis] * simulationData.restitution;
    }
}

__device__ void handlePeriodicBoundariesForAxis(ParticlesData particles,
                                                uint32_t id,
                                                const Simulation::Parameters& simulationData,
                                                int axis)
{
    const auto minBoundary = simulationData.domain.min[axis] + particles.radiuses[id];
    const auto maxBoundary = simulationData.domain.max[axis] - particles.radiuses[id];
    const auto domainLength =
        simulationData.domain.max[axis] - simulationData.domain.min[axis] - (2.F * particles.radiuses[id]);

    if (particles.positions[id][axis] < minBoundary)
    {
        // Move particle to the other end of the domain
        particles.positions[id][axis] += domainLength;
    }

    if (particles.positions[id][axis] > maxBoundary)
    {
        // Move particle to the beginning of the domain
        particles.positions[id][axis] -= domainLength;
    }
}

__global__ void computeExternalForces(ParticlesData particles, Simulation::Parameters simulationData, float deltaTime)
{
    const auto idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    // Apply gravity force to velocity
    particles.velocities[idx] += glm::vec4 {simulationData.gravity, 0.F} * deltaTime;
    // Special handling for Taylor-Green vortex - initialize velocity field
    if (simulationData.testCase == cuda::Simulation::Parameters::TestCase::TaylorGreenVortex)
    {
        // Only set velocity at the first time step - check if velocity is zero
        if (glm::length(glm::vec3(particles.velocities[idx])) < 1e-6F)
        {
            const auto pos = particles.positions[idx];
            const auto domainMin = simulationData.domain.min;
            const auto domainSize = simulationData.domain.max - domainMin;
            // Map position to [0, 2pi] range for Taylor-Green equations
            const auto x = ((pos.x - domainMin.x) / domainSize.x) * 2.0F * glm::pi<float>();
            const auto y = ((pos.y - domainMin.y) / domainSize.y) * 2.0F * glm::pi<float>();
            const auto z = ((pos.z - domainMin.z) / domainSize.z) * 2.0F * glm::pi<float>();
            // Calculate Taylor-Green velocity field
            const auto u = std::cos(x) * std::sin(y) * std::cos(z);
            const auto v = -std::sin(x) * std::cos(y) * std::cos(z);
            const auto w = 0.0F;

            // Set the velocity
            particles.velocities[idx] = glm::vec4(u, v, w, 0.0F);
        }
    }

    // Update predicted positions
    particles.predictedPositions[idx] = particles.positions[idx] + particles.velocities[idx] * deltaTime;
}

__global__ void countNeighbors(ParticlesData particles,
                               SphSimulation::State state,
                               Simulation::Parameters simulationData,
                               uint32_t* neighborCounts)
{
    const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }
    const auto position = particles.positions[idx];
    const auto radiusSquared = particles.smoothingRadiuses[idx] * particles.smoothingRadiuses[idx];
    uint32_t count = 0;

    forEachNeighbour(position,
                     particles,
                     simulationData,
                     state.grid,
                     [&](const auto neighbourIdx, const glm::vec4& adjustedPos) {
                         if (idx == neighbourIdx)
                         {
                             return;
                         }

                         // Use the adjustedPos for distance calculation
                         const auto offsetToNeighbour = adjustedPos - position;
                         const auto distanceSquared = glm::dot(offsetToNeighbour, offsetToNeighbour);

                         if (distanceSquared <= radiusSquared)
                         {
                             count++;
                         }
                     });

    neighborCounts[idx] = count;
}

__global__ void calculateDensityDeviations(ParticlesData particles, float restDensity)
{
    const auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= particles.particleCount)
    {
        return;
    }

    const auto deviation = (particles.densities[idx] - restDensity) / restDensity;
    particles.densityDeviations[idx] = deviation;
}
}
