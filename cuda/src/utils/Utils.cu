#include "hip/hip_runtime.h"
#include <hip/hip_vector_types.h>

#include <cstdint>
#include <glm/common.hpp>
#include <glm/ext/vector_float3.hpp>
#include <glm/ext/vector_float4.hpp>
#include <utility>

#include "Utils.cuh"
#include "cuda/Simulation.cuh"
#include "glm/ext/vector_uint3.hpp"
#include "simulation/adaptive/SphSimulation.cuh"

namespace sph::cuda
{

__constant__ int3 offsets[27] = {
    {0,  0,  0 },
    {0,  0,  -1},
    {0,  0,  1 },
    {0,  -1, 0 },
    {0,  -1, -1},
    {0,  -1, 1 },
    {0,  1,  0 },
    {0,  1,  -1},
    {0,  1,  1 },
    {-1, 0,  0 },
    {-1, 0,  -1},
    {-1, 0,  1 },
    {-1, -1, 0 },
    {-1, -1, -1},
    {-1, -1, 1 },
    {-1, 1,  0 },
    {-1, 1,  -1},
    {-1, 1,  1 },
    {1,  0,  0 },
    {1,  0,  -1},
    {1,  0,  1 },
    {1,  -1, 0 },
    {1,  -1, -1},
    {1,  -1, 1 },
    {1,  1,  0 },
    {1,  1,  -1},
    {1,  1,  1 }
};

//__device__ auto calculateCellIndex(glm::vec4 position,
//                                   const Simulation::Parameters& simulationData,
//                                   const SphSimulation::Grid& grid) -> glm::uvec3
//{
//    const auto relativePosition = glm::vec3 {position} - simulationData.domain.min;
//    const auto clampedPosition =
//        glm::clamp(relativePosition, glm::vec3(0.F), simulationData.domain.max - simulationData.domain.min);
//
//    return glm::uvec3 {clampedPosition.x / grid.cellSize.x,
//                       clampedPosition.y / grid.cellSize.y,
//                       clampedPosition.z / grid.cellSize.z};
//}

__device__ auto calculateCellIndex(glm::vec4 position,
                                   const Simulation::Parameters& simulationData,
                                   const SphSimulation::Grid& grid) -> glm::uvec3
{
    const auto relativePosition = glm::vec3 {position} - simulationData.domain.min;
    const auto domainSize = simulationData.domain.max - simulationData.domain.min;
    auto wrappedPosition = relativePosition;
    // Dla Poiseuille flow - tylko X jest periodic
    if (simulationData.testCase == Simulation::Parameters::TestCase::PoiseuilleFlow)
    {
        // Wrap X coordinate
        wrappedPosition.x = fmod(wrappedPosition.x, domainSize.x);
        if (wrappedPosition.x < 0)
        {
            wrappedPosition.x += domainSize.x;
        }

        // Clamp Y i Z
        wrappedPosition.y = glm::clamp(wrappedPosition.y, 0.0f, domainSize.y);
        wrappedPosition.z = glm::clamp(wrappedPosition.z, 0.0f, domainSize.z);
    }
    auto cellIndex = glm::uvec3 {static_cast<uint32_t>(wrappedPosition.x / grid.cellSize.x),
                                 static_cast<uint32_t>(wrappedPosition.y / grid.cellSize.y),
                                 static_cast<uint32_t>(wrappedPosition.z / grid.cellSize.z)};

    // Zabezpieczenie przed out-of-bounds
    cellIndex.x = glm::min(cellIndex.x, grid.gridSize.x - 1u);
    cellIndex.y = glm::min(cellIndex.y, grid.gridSize.y - 1u);
    cellIndex.z = glm::min(cellIndex.z, grid.gridSize.z - 1u);

    return cellIndex;
}

__device__ auto flattenCellIndex(glm::uvec3 cellIndex, glm::uvec3 gridSize) -> uint32_t
{
    return cellIndex.x + (cellIndex.y * gridSize.x) + (cellIndex.z * gridSize.x * gridSize.y);
}

__device__ auto getStartEndIndices(glm::uvec3 cellIndex, const SphSimulation::Grid& grid) -> std::pair<int32_t, int32_t>
{
    if (cellIndex.x >= grid.gridSize.x || cellIndex.y >= grid.gridSize.y || cellIndex.z >= grid.gridSize.z)
    {
        return {-1, -1};
    }

    const auto neighbourCellId = flattenCellIndex(cellIndex, grid.gridSize);

    return {grid.cellStartIndices[neighbourCellId], grid.cellEndIndices[neighbourCellId]};
}

}
